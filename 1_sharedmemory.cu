#include <cstdio>
#include <cstdlib>
#include <cmath>

#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#define CUDA_CALL(func) \
    do { \
      hipError_t err = (func); \
      if(err != hipSuccess) { \
        printf("CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
      } \
    } while(0)


// 每个thread block内线程为BLOCKSIZE * BLOCKSIZE
#define BLOCK_SIZE_M 32
#define BLOCK_SIZE_N 32
#define BLOCK_SIZE_K 32

#define VLX 32
#define VLY 32

template <int M, int N, int K>
__global__ void MatMulKernel_sharedMemory(float* A, float* B, float* C) {
  float value = 0.0;
  int row_block_start = blockIdx.y * BLOCK_SIZE_M;
  int col_block_start = blockIdx.x * BLOCK_SIZE_N;
  for (int i = 0; i < K / BLOCK_SIZE_K; ++i) {
    __shared__ float asub[BLOCK_SIZE_M][BLOCK_SIZE_K];
    __shared__ float bsub[BLOCK_SIZE_K][BLOCK_SIZE_N];
    for (int j = 0; j < BLOCK_SIZE_K / BLOCK_SIZE_N; ++j) {
      asub[threadIdx.y][j * BLOCK_SIZE_N + threadIdx.x] = A[(row_block_start + threadIdx.y) * K + i * BLOCK_SIZE_K + j * BLOCK_SIZE_N + threadIdx.x];
    }
    for (int j = 0; j < BLOCK_SIZE_K / BLOCK_SIZE_M; ++j) {
      bsub[j * BLOCK_SIZE_M + threadIdx.y][threadIdx.x] = B[(i * BLOCK_SIZE_K + j * BLOCK_SIZE_M + threadIdx.y) * N + col_block_start + threadIdx.x];
    }
    __syncthreads();
    for (int j = 0; j < BLOCK_SIZE_K; ++j) {
      value += asub[threadIdx.y][j] * bsub[j][threadIdx.x];
    }
    __syncthreads();
  }
  C[(row_block_start + threadIdx.y) * N + (col_block_start + threadIdx.x)] = value;
}

template <int M, int K, int N, bool enableProfiler>
float testMatMul(bool checkResult) {
  float* h_A, *h_B, *h_C;
  float* d_A, *d_B, *d_C;

  // 分配主机内存.
  h_A = (float*)malloc(M * K * sizeof(float));
  h_B = (float*)malloc(K * N * sizeof(float));
  h_C = (float*)malloc(M * N * sizeof(float));

  // 分配设备内存
  CUDA_CALL(hipMalloc((void**)&d_A, M * K * sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&d_B, K * N * sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&d_C, M * N * sizeof(float)));

  // 初始化主机内存
  for (int i = 0; i < M * K; ++i) {
    h_A[i] = rand() / (float)RAND_MAX;
  }
  for (int i = 0; i < K * N; ++i) {
    h_B[i] = rand() / (float)RAND_MAX;
  }

  // 将数据从主机内存拷贝到设备内存
  CUDA_CALL(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));

  // 创建CUDA事件
  hipEvent_t start, stop;
  CUDA_CALL(hipEventCreate(&start));
  CUDA_CALL(hipEventCreate(&stop));

  dim3 dimBlock(BLOCK_SIZE_N, BLOCK_SIZE_M);
  dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
  CUDA_CALL(hipEventRecord(start));
  if (enableProfiler) {
    hipProfilerStart();
  }
  MatMulKernel_sharedMemory<M, N, K><<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
      printf("Error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  CUDA_CALL(hipDeviceSynchronize());
  if (enableProfiler) {
    hipProfilerStop();
  }
  CUDA_CALL(hipEventRecord(stop));
  CUDA_CALL(hipEventSynchronize(stop));

  float milliseconds = 0;
  CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));

  // 清理
  free(h_A);
  free(h_B);
  free(h_C);
  CUDA_CALL(hipFree(d_A));
  CUDA_CALL(hipFree(d_B));
  CUDA_CALL(hipFree(d_C));
  CUDA_CALL(hipEventDestroy(start));
  CUDA_CALL(hipEventDestroy(stop));
  return milliseconds;
}

int main() {
  hipDeviceProp_t prop;
  int count;
  hipGetDeviceCount(&count);
  for (int i = 0; i < count; i++) {
    hipGetDeviceProperties(&prop, i);
    std::cout << "Device " << i << ":\n";
    std::cout << "Maximum threads per block: " << prop.maxThreadsPerBlock << "\n";
    std::cout << "Maximum dimension size of a thread block: (" << prop.maxThreadsDim[0] << ", " << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << ")\n";
    std::cout << "Maximum dimension size of a grid size: (" << prop.maxGridSize[0] << ", " << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << ")\n";
  }

  constexpr int times = 30;
  constexpr int M = 3840, K = 2880, N = 3840;
  constexpr bool checkResult = false, enableProfiler = false;
  float accMillis = 0.0;
  for (int i = 0; i <  times; ++i) {
    accMillis += testMatMul<M, K, N, enableProfiler>(checkResult);
    if (((i + 1) % 10) == 0) {
      printf("Testing process: %d / %d\n", (i + 1), times);
    }
  }
  printf("M=%d, K=%d, N=%d, bs_m=%d, bs_n=%d, bs_k=%d, enableProfiler=%d, MatMul: Totally elapsed time in GPU was %.2f ms, %.2f ms per operation\n",
                  M, K, N, BLOCK_SIZE_M, BLOCK_SIZE_N, BLOCK_SIZE_K, enableProfiler ? 1: 0, accMillis, accMillis / times);
}
